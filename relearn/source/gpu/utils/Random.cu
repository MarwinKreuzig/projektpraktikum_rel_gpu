#include "Random.cuh"

#include "../Commons.cuh"
#include "../neurons/NeuronsExtraInfos.cuh"

#include "hip/hip_runtime.h"
#include "hiprand.h"


namespace gpu::RandomHolder {

    __device__ int seed = 42;

    __device__ size_t number_neurons = 0;

    __device__ random_state_type init(const size_t step, size_t _number_neurons, const RandomKeyHolder key, const size_t neuron_id) {
        number_neurons = _number_neurons;
        random_state_type state;
        hiprand_init(seed + step, neuron_id, key, &state);
        return state;
    }

    __device__ void skip_to_next_item(hiprandState* state) {
        skipahead(number_neurons, state);
    }

    __device__ double get_percentage(hiprandState* state) {
        const auto value = hiprand_uniform_double(state);
        skip_to_next_item(state);
        return value;
    }

    __device__ double get_normal(hiprandState* state) {
        const auto value = hiprand_normal_double(state);
        skip_to_next_item(state);
        return value;
    }

    __device__ double get_normal(hiprandState* state, double mean, double stddev) {
        const auto value = get_normal(state) * stddev + mean;
        return value;
    }
};