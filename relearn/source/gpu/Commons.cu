#include "Commons.cuh"


inline void gpuAssert(hipError_t code, const char* file, int line, bool abort) {
    if (code == hipSuccess) {
        return;
    }

    fprintf(stderr, "GPUassert: %s %s %d\n", hipGetErrorString(code), file, line);
    if (abort) {
        exit(code);
    }
}

inline __device__ void deviceAssert(hipError_t code, const char* file, int line, bool abort) {
    if (code == hipSuccess) {
        return;
    }

    printf("GPU device assert: %s %s %d\n", hipGetErrorString(code), file, line);
    if (abort) {
        __trap();
    }
}

inline void cuda_memcpy_to_host(void* devPtr, void* hostPtr, size_t size_type, size_t number_elements) {
    hipDeviceSynchronize();
    gpu_check_last_error();
    hipMemcpy(hostPtr, devPtr, size_type * number_elements, hipMemcpyDeviceToHost);
    gpu_check_last_error();
    hipDeviceSynchronize();
    gpu_check_last_error();
}

inline void cuda_memcpy_to_device(void* devPtr, void* hostPtr, size_t size_type, size_t number_elements) {
    hipDeviceSynchronize();
    gpu_check_last_error();
    hipMemcpy(devPtr, hostPtr, size_type * number_elements, hipMemcpyHostToDevice);
    gpu_check_last_error();
    hipDeviceSynchronize();
    gpu_check_last_error();
}

inline void* cuda_malloc(size_t size, void* devPtr) {
    void* devPtrMalloc;
    hipDeviceSynchronize();
    gpu_check_last_error();
    hipMalloc(&devPtrMalloc, size);
    gpu_check_last_error();
    hipDeviceSynchronize();

    cuda_memcpy_to_device(devPtr, &devPtrMalloc, sizeof(void*), 1);

    return devPtrMalloc;
}

inline void* cuda_malloc(size_t size) {
    void* devPtrMalloc;
    hipDeviceSynchronize();
    gpu_check_last_error();
    hipMalloc(&devPtrMalloc, size);
    gpu_check_last_error();
    hipDeviceSynchronize();

    return devPtrMalloc;
}

inline __device__ void* device_malloc(size_t size) {
    // #warning Do not use malloc in device code
    void* devPtrMalloc;
    hipMalloc(&devPtrMalloc, size);
    return devPtrMalloc;
}

inline void* cuda_calloc(size_t size, void* devPtr) {
    void* devPtrMalloc = cuda_malloc(size, devPtr);
    hipMemset(devPtrMalloc, 0, size);
    gpu_check_last_error();
    return devPtrMalloc;
}

inline void* cuda_calloc(size_t size) {
    void* devPtrMalloc = cuda_malloc(size);
    hipMemset(devPtrMalloc, 0, size);
    gpu_check_last_error();
    return devPtrMalloc;
}

inline __device__ void* device_calloc(size_t size) {
    void* devPtrMalloc = device_malloc(size);
    hipMemsetAsync(devPtrMalloc, 0, size);
    return devPtrMalloc;
}

inline __device__ size_t block_thread_to_neuron_id(size_t block_id, size_t thread_id, size_t block_size) {
    return block_id * block_size + thread_id;
}

inline __device__ __host__ int get_number_blocks(int number_threads_per_block, int number_total_threads) {
    int number_blocks = number_total_threads / number_threads_per_block;
    if (number_total_threads % number_threads_per_block != 0) {
        number_blocks++;
    }

    if (number_blocks == 0) {
        return 1;
    }

    return number_blocks;
}