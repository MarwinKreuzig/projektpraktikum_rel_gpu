#include "hip/hip_runtime.h"
#include "BackgroundActivity.cuh"
#include "../../Commons.cuh"
#include <memory.h>

namespace gpu::background {
__device__ BackgroundActivity::BackgroundActivity() { }

__device__ double BackgroundActivity::get(size_t step, size_t neuron_id) {
    double b;
    if (extra_infos->disable_flags[neuron_id] == UpdateStatus::Disabled) {
        b = 0.0;
    } else {
        b = get_internal(step, neuron_id);
    }
    background_cache[neuron_id] = b;
    return b;
}

__device__ void BackgroundActivity::set_extra_infos(gpu::neurons::NeuronsExtraInfos* _extra_infos) {
    extra_infos = _extra_infos;
}

__device__ gpu::neurons::NeuronsExtraInfos* BackgroundActivity::get_extra_infos() {
    RelearnGPUException::device_check(extra_infos != nullptr, "BackgroundActivity::get_extra_infos: Pointer is null");
    return extra_infos;
}

__device__ Constant::Constant(double c)
    : BackgroundActivity()
    , constant(c) {
}

__device__ double Constant::get_internal(size_t step, size_t neuron_id) const {
    return constant;
}

__device__ Normal::Normal(double _mean, double _stddev)
    : BackgroundActivity()
    , mean(_mean)
    , stddev(_stddev) {
}

__global__ void update_input_for_all_neurons_kernel(gpu::background::BackgroundActivity* calculator, size_t step) {
    const auto neuron_id = block_thread_to_neuron_id(blockIdx.x, threadIdx.x, blockDim.x);

    if (neuron_id >= calculator->get_extra_infos()->num_neurons) {
        return;
    }

    const auto v = calculator->get(step, neuron_id);
}

BackgroundActivityHandleImpl::BackgroundActivityHandleImpl(void* calculator)
    : background_calculator(calculator) {
    _init();
}
void BackgroundActivityHandleImpl::init(size_t num_neurons) {
    background_cache.resize(num_neurons);
}

void BackgroundActivityHandleImpl::create_neurons(size_t num_neurons) {
    background_cache.resize(background_cache.get_size() + num_neurons);
}

std::vector<double> BackgroundActivityHandleImpl::get_background_activity() {
    std::vector<double> v;
    background_cache.copy_to_host(v);
    return v;
}

void BackgroundActivityHandleImpl::set_extra_infos(const std::unique_ptr<gpu::neurons::NeuronsExtraInfosHandle>& extra_infos_handle) {
    cuda_generic_kernel<<<1, 1>>>([=] __device__(BackgroundActivity * calculator, gpu::neurons::NeuronsExtraInfos * extra_infos) { calculator->set_extra_infos(extra_infos); }, (BackgroundActivity*)background_calculator, (neurons::NeuronsExtraInfos*)(extra_infos_handle.get()->get_device_pointer()));
}

void BackgroundActivityHandleImpl::update_input_for_all_neurons_on_gpu(RelearnGPUTypes::step_type step, RelearnGPUTypes::number_neurons_type number_local_neurons) {
    RelearnGPUException::check(number_local_neurons > 0, "BackgroundActivity::update_input_for_all_neurons_on_gpu: Number neurons is 0");
    RelearnGPUException::check(background_calculator != nullptr, "BackgroundActivity::update_input_for_all_neurons_on_gpu: Device pointer is null");

    const auto num_threads = get_number_threads(update_input_for_all_neurons_kernel, number_local_neurons);
    const auto num_blocks = get_number_blocks(num_threads, number_local_neurons);

    update_input_for_all_neurons_kernel<<<num_blocks, num_threads>>>((gpu::background::BackgroundActivity*)background_calculator, step);
    hipDeviceSynchronize();
    gpu_check_last_error();
}

void* BackgroundActivityHandleImpl::get_device_pointer() {
    return background_calculator;
}

void BackgroundActivityHandleImpl::_init() {
    void* background_cache_ptr = (void*)execute_and_copy<void*>([=] __device__(void* calculator) -> void* { return &((gpu::background::BackgroundActivity*)calculator)->background_cache; }, background_calculator);
    background_cache = gpu::Vector::CudaArrayDeviceHandle<double>(background_cache_ptr);
}

std::shared_ptr<BackgroundHandle> set_constant_background(double c) {
    void* background_calculator = init_class_on_device<gpu::background::Constant>(c);
    return std::make_shared<BackgroundActivityHandleImpl>(background_calculator);
}

std::shared_ptr<BackgroundHandle> set_normal_background(double mean, double stddev) {
    void* background_calculator = init_class_on_device<gpu::background::Normal>(mean, stddev);
    return std::make_shared<BackgroundActivityHandleImpl>(background_calculator);
}

std::shared_ptr<BackgroundHandle> set_fast_normal_background(double mean, double stddev, size_t multiplier) {
    void* background_calculator = init_class_on_device<gpu::background::Normal>(mean, stddev);
    return std::make_shared<BackgroundActivityHandleImpl>(background_calculator);
}

};